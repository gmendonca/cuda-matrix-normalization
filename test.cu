

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();

/* returns a seed for srand based on the time */
unsigned int time_seed() {
  struct timeval t;
  struct timezone tzdummy;

  gettimeofday(&t, &tzdummy);
  return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
  int seed = 0;  /* Random seed */
  char uid[32]; /*User name */

  /* Read command-line arguments */
  srand(time_seed());  /* Randomize */

  if (argc == 3) {
    seed = atoi(argv[2]);
    srand(seed);
    printf("Random seed = %i\n", seed);
  }
  if (argc >= 2) {
    N = atoi(argv[1]);
    if (N < 1 || N > MAXN) {
      printf("N = %i is out of range.\n", N);
      exit(0);
    }
  }
  else {
    printf("Usage: %s <matrix_dimension> [random seed]\n",
           argv[0]);
    exit(0);
  }

  /* Print parameters */
  printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
  int row, col;

  printf("\nInitializing...\n");
  for (col = 0; col < N; col++) {
    for (row = 0; row < N; row++) {
      A[row][col] = (float)rand() / 32768.0;
      B[row][col] = 0.0;
    }
  }

}

/* Print input matrices */
void print_inputs() {
  int row, col;

  if (N < 10) {
    printf("\nA =\n\t");
    for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
	    printf("%5.2f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
      }
    }
  }
}

void print_B() {
    int row, col;

    if (N < 10) {
        printf("\nB =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%1.10f%s", B[row][col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
	    hipGetErrorString(err),__FILE__,__LINE__);	\
    exit(-1);						\
  }                                                     \

__global__ void vecAdd (float* d_A, float* d_B, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
    d_B[i] = 1.5;
}

int main(int argc, char **argv) {

    /* Timing variables */
    struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    clock_t etstart2, etstop2;  /* Elapsed times using times() */
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;  /* CPU times for my processes */

    /* Process program parameters */
    parameters(argc, argv);

    /* Initialize A and B */
    initialize_inputs();

    /* Print input matrices */
    print_inputs();

    /* Start Clock */
    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    etstart2 = times(&cputstart);

  hipError_t err;

  // Memory allocation for h_A, h_B and h_C (in the host)
  float h_A[N];
  int j = 0;
  for (; j < 10; j++)
     h_A[j] = j + 1.5;
  float h_B[N];

  float *d_A, *d_B;

  // Memory allocation for d_A, d_B and d_C (in the device)
  err = hipMalloc((void **) &d_A, sizeof(float)*N);
  CHECK_ERR(err);

  err =hipMalloc((void **) &d_B, sizeof(float)*N);
  CHECK_ERR(err);

  // Copying memory to device
  err = hipMemcpy(d_A, h_A, sizeof(float)*N, hipMemcpyHostToDevice);
  CHECK_ERR(err);


  // Calling the kernel
  vecAdd<<<ceil(N/256.0), 256>>>(d_A,d_B,N);

  // Copying results back to host
  err = hipMemcpy(h_B, d_B, sizeof(float)*N, hipMemcpyDeviceToHost);
  CHECK_ERR(err);

  int i = 0;
  for (; i < N; i++)
    printf("h_B[%d] = %f\n",i,h_B[i]);

    /* Stop Clock */
    gettimeofday(&etstop, &tzdummy);
    etstop2 = times(&cputstop);
    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    /* Display output */
    print_B();

    /* Display timing results */
    printf("\nElapsed time = %g ms.\n",
  	 (float)(usecstop - usecstart)/(float)1000);

    printf("(CPU times are accurate to the nearest %g ms)\n",
  	 1.0/(float)CLOCKS_PER_SEC * 1000.0);
    printf("My total CPU time for parent = %g ms.\n",
  	 (float)( (cputstop.tms_utime + cputstop.tms_stime) -
  		  (cputstart.tms_utime + cputstart.tms_stime) ) /
  	 (float)CLOCKS_PER_SEC * 1000);
    printf("My system CPU time for parent = %g ms.\n",
  	 (float)(cputstop.tms_stime - cputstart.tms_stime) /
  	 (float)CLOCKS_PER_SEC * 1000);
    printf("My total CPU time for child processes = %g ms.\n",
  	 (float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
  		  (cputstart.tms_cutime + cputstart.tms_cstime) ) /
  	 (float)CLOCKS_PER_SEC * 1000);
        /* Contrary to the man pages, this appears not to include the parent */
    printf("--------------------------------------------\n");

    exit(0);

}